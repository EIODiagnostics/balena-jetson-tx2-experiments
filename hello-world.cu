// source https://www.computer-graphics.se/hello-world-for-cuda.html
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA
// with an array of offsets. Then the offsets are added in parallel
// to produce the string "World!"
// By Ingemar Ragnemalm 2010

// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <unistd.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void hello(char *a, int *b) 
{
	int idx = threadIdx.x; // + 6;
	if (idx < N)
		a[idx] = a[threadIdx.x] + b[threadIdx.x];
}

void exitWithFailure(hipError_t err, int line) 
{
	printf("Error %d at line %d\n", err, line);
	exit(1);
}

int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);
	hipError_t err;

	err = hipMalloc( (void**)&ad, csize ); 
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipMalloc( (void**)&bd, isize ); 
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipFree( ad );
	if (err != hipSuccess) exitWithFailure(err, __LINE__);
	err = hipFree( bd );
	
	printf("%s\n", a);
	return EXIT_SUCCESS;
}
